#include<stdio.h>
//#include<cuda.h>
#include<hip/hip_runtime.h>
#define BLOCK_SIZE 16
#define GRID_SIZE 1


__global__ void test(int A[BLOCK_SIZE][BLOCK_SIZE], int B[BLOCK_SIZE][BLOCK_SIZE],int C[BLOCK_SIZE][BLOCK_SIZE])
{

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < BLOCK_SIZE && j < BLOCK_SIZE)
        C[i][j] = A[i][j] + B[i][j];

}

int main()
{


    int d_A[BLOCK_SIZE][BLOCK_SIZE];
    int d_B[BLOCK_SIZE][BLOCK_SIZE];
    int d_C[BLOCK_SIZE][BLOCK_SIZE];

    int C[BLOCK_SIZE][BLOCK_SIZE];

    for(int i=0;i<BLOCK_SIZE;i++)
      for(int j=0;j<BLOCK_SIZE;j++)
      {
        d_A[i][j]=i+j;
        d_B[i][j]=i+j;
      }


    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid(GRID_SIZE, GRID_SIZE); 

    test<<<dimGrid, dimBlock>>>(d_A,d_B,d_C); 

    hipMemcpy(C,d_C,BLOCK_SIZE*BLOCK_SIZE , hipMemcpyDeviceToHost);

for(int i=0;i<BLOCK_SIZE;i++)
      for(int j=0;j<BLOCK_SIZE;j++)
      {
        printf("%d\n",C[i][j]);

      }
}

