#include<stdio.h>
//#include<cuda.h>
#include<hip/hip_runtime.h>



#define N 4
#define BLOCK_DIM 4
__global__ void matrixAdd (int *dev_a);
int main() {
int a[N*N]={};
int i;
for(i=0;i<16;i++)
{
  printf("Enter the %dth element= ",i);
 // a[i]=i*2;
   scanf("%d",&a[i]);
}
int *dev_a;
//int dev_b;
int size = N * N * sizeof(int);
// initialize a and b with real values (NOT SHOWN)
hipMalloc((void**)&dev_a, size);
//cudaMalloc((void**)&dev_b, size);
//cudaMalloc((void**)&dev_c, size);
hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
//cudaMemcpy(dev_b, b, size, cudaMemcpyHostToDevice);
dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(N/dimBlock.y));
matrixAdd<<<dimGrid,dimBlock>>>(dev_a);
hipMemcpy(a, dev_a, size, hipMemcpyDeviceToHost);
hipFree(dev_a); 
//cudaFree(dev_b); 

}
__global__ void matrixAdd (int *dev_a) {
int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;
int index = col + row * N;
//dev_b=index;
//}

if (col < N && row < N) {
//c[index] = a[index] + b[index];
printf("%d\n",dev_a[index]);

}
}
